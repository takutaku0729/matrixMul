#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling
 * approach. It has been written for clarity of exposition to illustrate various
 * CUDA programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication. See also: V. Volkov and
 * J. Demmel, "Benchmarking GPUs to tune dense linear algebra," in Proc. 2008
 * ACM/IEEE Conf. on Supercomputing (SC '08), Piscataway, NJ: IEEE Press, 2008,
 * pp. Art. 31:1-11.
 */

// System includes
#include <assert.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <fstream>
#include <random>
#include <unistd.h>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include "matrixMul.h"

#define INF INFINITY
#define BLOCK_SIZE 32
#define BLOCK_NUM 512
#define INFP 100
#define ZEROP 0
#define MODE 3   // tropical = 1, else = 2, infskip = 3, zeroskip = 4
#define ADD_MODE 1 //1:min-plus 2:max-plus
#define MAXSIZE 16384
#define LOOP 1
#define FILENAME USA-road-d.NY.gr

//define for switching debug mode
//#define DEBUG



/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */

__global__ void MatrixMulCUDA(float* C, float* A, float* B, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

__global__ void MinPlusTrop(float* C, float* A, float* B, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub = (Csub <= (As[ty][k] + Bs[k][tx]) ? Csub : As[ty][k] + Bs[k][tx]);
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the blocsub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

__global__ void MinPlusTropSkip(float* C, float* A, float* B, float* infA, float* infB, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = INF;

    int indexConstA = wA * ty + tx;
    int indexConstB = wB * ty + tx;

    int infIndexConstA = (wA / BLOCK_SIZE) * by;
    int infIndexConstB = wB / BLOCK_SIZE;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin, infIndexA = infIndexConstA, infIndexB = bx; a <= aEnd; a += aStep, b += bStep, infIndexA++, infIndexB += infIndexConstB) {
        //skip execution
        if ((infA[infIndexA] == 1) || (infB[infIndexB] == 1)) {
            continue;
        }

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + indexConstA];
        Bs[ty][tx] = B[b + indexConstB];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub = (Csub <= (As[ty][k] + Bs[k][tx]) ? Csub : As[ty][k] + Bs[k][tx]);
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the blocsub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + indexConstB] = Csub;
}

__global__ void MinPlusTropZeroSkip(float* C, float* A, float* B, float* infA, float* infB, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = INF;

    int indexConstA = wA * ty + tx;
    int indexConstB = wB * ty + tx;

    int infIndexConstA = (wA / BLOCK_SIZE) * by;
    int infIndexConstB = wB / BLOCK_SIZE;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin, infIndexA = infIndexConstA, infIndexB = bx; a <= aEnd; a += aStep, b += bStep, infIndexA++, infIndexB += infIndexConstB) {
        //skip execution
        if ((infA[infIndexA] == 1) || (infB[infIndexB] == 1)) {
            continue;
        }

        if ((infA[infIndexA] == -1) && (infB[infIndexB] == -1)) {
            Csub = 0;
            break;
        }

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + indexConstA];
        Bs[ty][tx] = B[b + indexConstB];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            if (Csub >= (As[ty][k] + Bs[k][tx])) {
                Csub = As[ty][k] + Bs[k][tx];
                if (Csub == 0) {
                    break;
                }
            }
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the blocsub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + indexConstB] = Csub;

}

__global__ void MaxPlusTrop(float* C, float* A, float* B, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub = (Csub >= (As[ty][k] + Bs[k][tx]) ? Csub : As[ty][k] + Bs[k][tx]);
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the blocsub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

__global__ void MaxPlusTropSkip(float* C, float* A, float* B, float* infA, float* infB, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = INF;

    int indexConstA = wA * ty + tx;
    int indexConstB = wB * ty + tx;

    int infIndexConstA = (wA / BLOCK_SIZE) * by;
    int infIndexConstB = wB / BLOCK_SIZE;
    

    //skip execution
    for (int infIndexA = infIndexConstA, infIndexB =bx; infIndexA < infIndexConstA + (wA / BLOCK_SIZE); infIndexA++, infIndexB += infIndexConstB) {
        if ((isinf(infA[infIndexA]) == 1) || (isinf(infB[infIndexB]) == 1)) {
            int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
            C[c + indexConstB] = INF;
            return;
        }
    }

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin, infIndexA = infIndexConstA, infIndexB = bx; a <= aEnd; a += aStep, b += bStep, infIndexA++, infIndexB += infIndexConstB) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + indexConstA];
        Bs[ty][tx] = B[b + indexConstB];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub = (Csub >= (As[ty][k] + Bs[k][tx]) ? Csub : As[ty][k] + Bs[k][tx]);
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the blocsub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + indexConstB] = Csub;
}

__global__ void InfCheck(float* C, float* A, float* B, float* infA, float* infB, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the first sub-matrix of A processed by the block
    int bBegin = wB * BLOCK_SIZE * by;

    __shared__ int infcheckA;
    __shared__ int infcheckB;

    infcheckA = 0;
    infcheckB = 0;

    __syncthreads();


    if (isinf(A[aBegin + tx + ty * wA]) == 0) {
        infcheckA = 1;
    }

    if (isinf(B[bBegin + tx + ty * wB]) == 0) {
        infcheckB = 1;
    }

    __syncthreads();
    
    if (tx == 0 && ty == 0) {
        if (infcheckA == 0) {
            infA[bx + (wA / BLOCK_SIZE) * by] = 1;
        }
        if (infcheckB == 0) {
            infB[bx + (wB / BLOCK_SIZE) * by] = 1;
        }
    }
}

__global__ void ZeroCheck(float* C, float* A, float* B, float* infA, float* infB, int wA, int wB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the first sub-matrix of A processed by the block
    int bBegin = wB * BLOCK_SIZE * by;

    __shared__ int zerocheckA;
    __shared__ int zerocheckB;

    zerocheckA = 0;
    zerocheckB = 0;

    __syncthreads();


    if (A[aBegin + tx + ty * wA] != 0) {
        zerocheckA = 1;
    }

    if (B[bBegin + tx + ty * wB] != 0) {
        zerocheckB = 1;
    }

    __syncthreads();

    if (tx == 0 && ty == 0) {
        if (zerocheckA == 0) {
            infA[bx + (wA / BLOCK_SIZE) * by] = -1;
        }
        if (zerocheckB == 0) {
            infB[bx + (wB / BLOCK_SIZE) * by] = -1;
        }
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
float MatrixMultiply(int argc, char** argv, int block_size, const dim3& dimsA,
    const dim3& dimsB, std::ofstream& writing_file, int mode, int addmode) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  unsigned int size_infA = (dimsA.x/BLOCK_SIZE) * (dimsA.y/BLOCK_SIZE);
  unsigned int mem_size_infA = sizeof(float) * size_infA;
  float *h_infA;
  checkCudaErrors(hipHostMalloc(&h_infA, mem_size_infA));
  unsigned int size_infB = (dimsB.x / BLOCK_SIZE) * (dimsB.y / BLOCK_SIZE);
  unsigned int mem_size_infB = sizeof(float) * size_infA;
  float *h_infB;
  checkCudaErrors(hipHostMalloc(&h_infB, mem_size_infB));
  
  hipStream_t stream;

  // Initialize host memory

  int InfP = INFP;  //set inf percentage
  int ZeroP = ZEROP;
  
#ifdef DEBUG
  ConstantInit(h_A, size_A, INF);
  ConstantInit(h_B, size_B, INF);
  //SetFileData(h_A, dimsA.x);    //set file data to host memory
  //SetFileData(h_B, dimsB.x);

  #else

  //set random (sparce) data to host memory

  ConstantInitRand(h_A, InfP, ZeroP, dimsA, 0);

  ConstantInitRand(h_B, InfP, ZeroP, dimsB, 0);

#endif

  ConstantInit(h_infA, size_infA, 0);  //init inf check matrix
  ConstantInit(h_infB, size_infB, 0);

  // Allocate device memory
  float* d_A, * d_B, * d_C, * inf_A, * inf_B;

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&inf_A), mem_size_infA));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&inf_B), mem_size_infB));
  // Allocate CUDA events that we'll use for timing
  hipEvent_t startA, stopA, startB, stopB;
  checkCudaErrors(hipEventCreate(&startA));
  checkCudaErrors(hipEventCreate(&stopA));
  checkCudaErrors(hipEventCreate(&startB));
  checkCudaErrors(hipEventCreate(&stopB));

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // copy host memory to device
  checkCudaErrors(
      hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(inf_A, h_infA, mem_size_infA, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(inf_B, h_infB, mem_size_infB, hipMemcpyHostToDevice, stream));

  // Setup execution parameters
  dim3 block(block_size, block_size);
  dim3 grid(dimsB.x / block.x, dimsA.y / block.y);

  // Create and start timer
  //printf("Computing result using CUDA Kernel...\n");

  checkCudaErrors(hipStreamSynchronize(stream));

  // Execute the kernel
  int nIter = 1;
  
  if (addmode == 1) {
      if (mode == 1) {
          checkCudaErrors(hipEventRecord(startA, stream));
          for (int j = 0; j < nIter; j++) {
              MinPlusTrop
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
          }
      }
      else if (mode == 2) {
          checkCudaErrors(hipEventRecord(startA, stream));
          for (int j = 0; j < nIter; j++) {
              MatrixMulCUDA
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
          }
      }
      else if (mode == 3) {
          for (int j = 0; j < nIter; j++) {
              checkCudaErrors(hipEventRecord(startB, stream));
              InfCheck
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
              checkCudaErrors(hipDeviceSynchronize());
              checkCudaErrors(hipEventRecord(stopB, stream));
              checkCudaErrors(hipEventRecord(startA, stream));
              MinPlusTropSkip
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
          }
      }
      else{
          for (int j = 0; j < nIter; j++) {
              checkCudaErrors(hipEventRecord(startB, stream));
              InfCheck
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
              checkCudaErrors(hipDeviceSynchronize());
              ZeroCheck
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
              checkCudaErrors(hipDeviceSynchronize());
              checkCudaErrors(hipEventRecord(stopB, stream));
              checkCudaErrors(hipEventRecord(startA, stream));
              MinPlusTropZeroSkip
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
          }
      }
  }
  else {
      if (mode == 1) {
          checkCudaErrors(hipEventRecord(startA, stream));
          for (int j = 0; j < nIter; j++) {
              MaxPlusTrop
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
          }
      }
      else if (mode == 2) {
          checkCudaErrors(hipEventRecord(startA, stream));
          for (int j = 0; j < nIter; j++) {
              MatrixMulCUDA
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
          }
      }
      else if (mode == 3) {
          checkCudaErrors(hipEventRecord(startA, stream));
          for (int j = 0; j < nIter; j++) {
              checkCudaErrors(hipEventRecord(startB, stream));
              InfCheck
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
              checkCudaErrors(hipDeviceSynchronize());
              checkCudaErrors(hipEventRecord(stopB, stream));
              checkCudaErrors(hipEventRecord(stopB, stream));
              MaxPlusTropSkip
                  << <grid, block, 0, stream >> > (d_C, d_A, d_B, inf_A, inf_B, dimsA.x, dimsB.x);
          }
      }
  }

  // Record the stop event
  checkCudaErrors(hipEventRecord(stopA, stream));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stopA));

  float msecTotalA = 0.0f;
  float msecTotalB = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotalA, startA, stopA));

  float msecTotal = msecTotalA;

  if (mode >= 3) {
      checkCudaErrors(hipEventElapsedTime(&msecTotalB, startB, stopB));
      msecTotal += msecTotalB;
  }

  // Copy result from device to host
  checkCudaErrors(
      hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(
      hipMemcpyAsync(h_infA, inf_A, mem_size_infA, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // Clean up memory
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipFree(inf_B));
  checkCudaErrors(hipFree(inf_A));
  checkCudaErrors(hipEventDestroy(startA));
  checkCudaErrors(hipEventDestroy(stopA));
  checkCudaErrors(hipEventDestroy(startB));
  checkCudaErrors(hipEventDestroy(stopB));

  return msecTotal;
}

/**
 * Program main
 */
int main(int argc, char **argv) {

    int infp = INFP;
    int zerop = ZEROP;
    int mode = MODE;
    int addmode = ADD_MODE;

    float matrix_result = 0;
    int max_size = MAXSIZE;
    int avg_count = LOOP;

    int opt;

    // getopt 

    opterr = 0;
    while ((opt = getopt(argc, argv, "m:s:i:z:l:")) != -1){
        switch(opt){
            case 'm':
                mode = stoi(optarg);
                break;
            
            case 's':
                max_size = stoi(optarg);
                break;
            
            case 'i':
                infp = stoi(optarg);
                break;
            
            case 'z':
                zerop = stoi(optarg);
                break;
            
            case 'l':
                avg_count = stoi(optarg);
                break;

            default:
                printf("unknown option\n");
                break;
        }
    }

    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    int block_size = BLOCK_SIZE;

    int block_num = BLOCK_NUM;

    dim3 dimsA(block_num * block_size, block_num * block_size, 1);
    dim3 dimsB(block_num * block_size, block_num * block_size, 1);

    #ifdef DEBUG //for debug

    int size = block_size * block_num;
    dimsA.x = size;
    dimsA.y = size;
    dimsB.x = size;
    dimsB.y = size;
    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x,
        dimsB.y);
        matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB, writing_file, MODE, ADD_MODE);
    exit(matrix_result);


    #else


    std::ofstream writing_file;
    std::string filename;
    if (mode == 4) {
        filename = "inf" + std::to_string(infp) + "zero" + std::to_string(zerop) + ".csv";
    }
    else {
        filename = "inf" + std::to_string(infp) + ".csv";
    }

    writing_file.open(filename, std::ios::out);

    printf("noskip\n");

    for (int size = block_size; size <= max_size; size *= 2) {
        writing_file << "noskip-" + std::to_string(size);
        dimsA.x = size;
        dimsA.y = size;
        dimsB.x = size;
        dimsB.y = size;
        float sum = 0;
        float sum_time = 0;
        double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
        static_cast<double>(dimsA.y) *static_cast<double>(dimsB.x);

        printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x,
            dimsB.y);
        for (int i = 0; i <= avg_count; i++) {
            matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB, writing_file, 1, addmode);
            float msecPerMatrixMul = (matrix_result) / nIter;
            double gigaFlops =
                (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
            writing_file << "," + std::to_string(gigaFlops);
            sum_time += matrix_result;
            sum += gigaFlops;
            }
            printf(
                    "Performance= %.2f GFlop/s, Time = %.3f msec, Size= %.0f Ops\n",
                    sum/avg_count, sum_time/avg_count, flopsPerMatrixMul);
        writing_file << "\n";
    }
    

    printf("\n\nskip\n");


    for (int size = block_size; size <= max_size; size *= 2) {
        writing_file << "skip-" + std::to_string(size);
        dimsA.x = size;
        dimsA.y = size;
        dimsB.x = size;
        dimsB.y = size;
        float sum = 0;
        float sum_time = 0;
        double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
        static_cast<double>(dimsA.y) *static_cast<double>(dimsB.x);

        printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x,
            dimsB.y);
        for (int i = 0; i <= avg_count; i++) {
            matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB, writing_file, mode, addmode);
            float msecPerMatrixMul = (matrix_result) / nIter;
            double gigaFlops =
                (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
            writing_file << "," + std::to_string(gigaFlops);
            sum_time += matrix_result;
            sum += gigaFlops;
            }
                printf(
                    "Performance= %.2f GFlop/s, Time = %.3f msec, Size= %.0f Ops\n",
                    sum/avg_count, sum_time/avg_count, flopsPerMatrixMul);
        writing_file << "\n";
    }
    exit(matrix_result);
    #endif

}
